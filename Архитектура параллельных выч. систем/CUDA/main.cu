#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "main.h"
#include "cuda.hpp"
#include "cuda_runtime.hpp"

// матрица состояний пластины
double p[I_MAX + 1][J_MAX + 1][K_MAX + 1];

void substract_rows_Gauss(int i_from, int i_to, double* A, int size, double* fr) {
    double coeff = A[size * i_to + i_from];
    for (int j = 0; j < size; j++) {
        A[size * i_to + j] -= A[size * i_from + j] * coeff;
    }
    fr[i_to] -= fr[i_from] * coeff;
}

// метод Гаусса
__global__ void Gauss(double* A, int size, double* fr) {
    double coeff;

    // forward
    for (int i = 0; i < size - 1; i++) {
        // > ---- make diag one ----
        coeff = A[i * size + i];
        for (int j = 0; j < size; j++) {
            A[i * size + j] /= coeff;
        }
        fr[i] /= coeff;
        // < ---- make diag one ----

        for (int k = i + 1; k < size; k++) {            
            // > ---- substract rows ----
            double coeff = A[size * k + i];
            for (int j = 0; j < size; j++) {
                A[size * k + j] -= A[size * i + j] * coeff;
            }
            fr[k] -= fr[i] * coeff;
            // < ---- substract rows ----
        }
    }
    // > ---- make diag one ----
    coeff = A[(size - 1) * size + (size - 1)];
    for (int j = 0; j < size; j++) {
        A[(size - 1) * size + j] /= coeff;
    }
    fr[(size - 1)] /= coeff;
    // < ---- make diag one ----

    // backward
    for (int i = size - 1; i > 0 ; i--) {
        for (int k = i - 1; k >= 0; k--) {
            // > ---- substract rows ----
            double coeff = A[size * k + i];
            for (int j = 0; j < size; j++) {
                A[size * k + j] -= A[size * i + j] * coeff;
            }
            fr[k] -= fr[i] * coeff;
            // < ---- substract rows ----
        }
    }
}

// напечатать состояние
void print_state(int k) {
    printf("Матрица p:\n");
    for (int i = 0; i <= I_MAX; i++) {
        for (int j = 0; j <= J_MAX; j++) {
            printf("%8.3lf ", p[i][j][k]);
        }
        printf("\n");
    }
    printf("\n");
}

// задать начальные условия
void begin_state() {
    // внутренние узлы из НУ
    for (int i = 1; i < I_MAX; i++) {
        for (int j = 1; j < J_MAX; j++) {
            p[i][j][0] = T_0;
        }
    }

    // верхняя и нижняя границы из ГУ
    for (int j = 0; j <= J_MAX; j++) {
        p[I_MAX][j][0] = T_BOTTOM();
        p[0][j][0] = T_UP();
    }

    // левая и правая границы из ГУ
    for (int i = 1; i <= I_MAX; i++) {
        p[i][0][0] = T_LEFT(p[i][1][0]);
        p[i][J_MAX][0] = T_RIGHT();
    }

}

void print_matrix(double* A, int rows, int cols, char* name) {
    printf("%s\n", name);
    /*for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%5.2lf ", A[cols * i + j]);
        }
        printf("\n");
    }*/
    for (int i = 0; i < rows * cols; i++) {
        printf("%lf\n", A[i]);
    }
    printf("\n");
}

void prepare_state(double* C, double* A, int size, int cmax, int k) {
    // Задание граничных условий в матрице

    // верх и низ
    // граничное условие первого рода
    for (int m = 0; m <= J_MAX; m++) {
        A[m * size + m] = 1;
        C[m] = T_UP();

        A[(cmax - m) * size + (cmax - m)] = 1;
        C[cmax - m] = T_BOTTOM();
    }

    // левая граница - каждый J_MAX + 1 -ый элемент
    // граничное условие второго рода
    for (int m = 0; m <= cmax; m += J_MAX + 1) {
        A[m * size + m] = 1;
        A[m * size + (m + 1)] = -1;
        C[m] = 0;
    }

    // правая граница - каждый J_MAX -ый элемент
    // граничное условие первого рода
    for (int m = J_MAX; m <= cmax; m += J_MAX + 1) {
        A[m * size + m] = 1;
        C[m] = T_RIGHT();
    }

    // неявная разностная схема
    // пропускаем все граничные условия по верху и первое по левой границе
    // в условии - останавливаемся до последней правой границы

    // коэффициенты
    double Tij_coef = D_X*D_X*D_Y*D_Y + 2*D_Y*D_Y*D_T + 2*D_X*D_X*D_T;
    double Ti_jv_coef = -D_Y*D_Y*D_T;
    double Tiv_j_coef = -D_X*D_X*D_T;
    double T_prev_coeff = D_Y*D_Y*D_X*D_X;

    for (int q = J_MAX + 2; q < cmax - J_MAX; q += J_MAX + 1 ) {
        for (int m = 0; m < J_MAX - 1; m++) {
            A[size * (q+m) + q+m ] = Tij_coef;                  // T i,j
            A[size * (q+m) + q+m+J_MAX+1] = Tiv_j_coef;         // T i+1,j
            A[size * (q+m) + q+m-J_MAX-1] =Tiv_j_coef;          // T i-1,j
            A[size * (q+m) + q+m+1] = Ti_jv_coef;               // T i,j+1
            A[size * (q+m) + q+m-1] = Ti_jv_coef;               // T i,j-1
            C[q+m] = p[(q+m)/(J_MAX+1)][(q+m)%(J_MAX+1)][k-1]*T_prev_coeff;
        }
    }

}

// рассчитать состояние
void count_state(int k) {
    // ----------------- CUDA 1. Выделение памяти хоста и заполнение ее -----------------

    // расчет начальных условий
    if (k == 0) {
        begin_state();
        return;
    }

    // создание исходных матриц СЛАУ
    int size = (I_MAX + 1) * (J_MAX + 1);
    int cmax = size - 1;    // максимальный номер строки в векторе с

    double* C = (double*)malloc(sizeof(double) * size);
    double* A = (double*)malloc(sizeof(double) * size * size);

    prepare_state(C, A, size, cmax, k);

    // ----------------- CUDA 2. Выделение памяти девайса и заполнение ее -----------------
    double *dC, *dA, *dsize; 
    hipMalloc((void**)(&dC), sizeof(double) * size);
    hipMalloc((void**)(&dA), sizeof(double) * size * size);
    hipMalloc((void**)(&dsize), sizeof(int));

    // --------------------- CUDA 3. Перенос данных из хоста девайсу ---------------------
    hipMemcpy((void*)dC, (void*)C, sizeof(double) * size, cudaMemcpyToDevice);
    hipMemcpy((void*)dA, (void*)A, sizeof(double) * size * size, cudaMemcpyToDevice);
    hipMemcpy((void*)dsize, (void*)(&size)), sizeof(int));

    // ----------------------- CUDA 4. Вызов ядра для решения СЛАУ -----------------------
    Gauss<<<1,1>>>(dA, dsize, dC);
    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();


    // --------------------- CUDA 5. Перенос данных из девайса хосту ---------------------
    hipMemcpy((void*)C, (void*)dC, sizeof(double) * size, cudaMemcpyToHost);

    // перенос значений из вектора C в матрицу p
    int gi = 0;
    for (int i = 0; i < I_MAX + 1; i++) {
        for (int j = 0; j < J_MAX + 1; j++) {
            p[i][j][k] = C[gi];
            gi++;
        }
    }

    // ------------------------ CUDA 6. Очистка всех видов памяти ------------------------
    hipFree(dC);
    hipFree(dA);
    hipFree(dsize);

    free(C);
    free(A);
}

int main () {
    for (int k = 0; k <= K_MAX; k++) {
        count_state(k);
        print_state(k);
    }  

    return 0;
}
